#include "hip/hip_runtime.h"
/*
This Source Code Form is subject to the terms of the Mozilla Public
License, v. 2.0. If a copy of the MPL was not distributed with this
file, You can obtain one at http://mozilla.org/MPL/2.0/. 
*/

/*!
\file examples/Grenaille/ssgls.cu
\brief Screen space GLS using c++/CUDA

\author: Nicolas Mellado, Gautier Ciaudo
*/

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cmath>
#include <algorithm>
#include <FreeImagePlus.h>
#include <vector>

#include "Eigen/Core"
#include "Patate/grenaille.h"

#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
	fprintf(stderr, "Error %s at line %d in file %s\n",					\
	hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
	exit(1);															\
	} }

//! [mypoint]
class MyPoint
{
public:
	enum {Dim = 3};
	typedef float Scalar;
	typedef Eigen::Matrix<Scalar, Dim, 1> VectorType;
	typedef Eigen::Matrix<Scalar, Dim, Dim> MatrixType;
	typedef Eigen::Matrix<Scalar, 2, 1>   ScreenVectorType;

	MULTIARCH inline MyPoint(const VectorType &pos        = VectorType::Zero(),
		const VectorType& normal     = VectorType::Zero(),
		const ScreenVectorType& spos = ScreenVectorType::Zero(),
		const Scalar dz = 0.f)
		: _pos(pos), _normal(normal), _spos(spos), _dz(dz){}

	MULTIARCH inline const VectorType& pos()	const { return _pos; }
	MULTIARCH inline const VectorType& normal()	const { return _normal; }
	MULTIARCH inline const ScreenVectorType& spos() const { return _spos; }
	MULTIARCH inline const float & dz()	const { return _dz; }


	MULTIARCH inline VectorType& pos()	 { return _pos; }
	MULTIARCH inline VectorType& normal()	 { return _normal; }
	MULTIARCH inline ScreenVectorType& spos() { return _spos; }
	MULTIARCH inline float& dz()	 { return _dz; }


private:
	ScreenVectorType _spos;
	VectorType	_pos, _normal;
	float _dz; // depth threshold
};
//! [mypoint]

typedef MyPoint::Scalar Scalar;
typedef MyPoint::VectorType VectorType;
typedef MyPoint::ScreenVectorType ScreenVectorType;

//! [w_def]
class ProjectWeightFunc: public Grenaille::DistWeightFunc<MyPoint, Grenaille::SmoothWeightKernel<Scalar> >{
public:
	typedef MyPoint::Scalar Scalar;
	typedef MyPoint::VectorType VectorType;

	/*
	Default constructor (needed by Grenaille). Note that the screenspace
	evaluation position is specified as parameter
	*/
	MULTIARCH inline ProjectWeightFunc(const Scalar& t                = 1.f,
		const ScreenVectorType &refPos = ScreenVectorType::Zero(),
		const Scalar &dz               = 0.f)
		: Grenaille::DistWeightFunc<MyPoint, Grenaille::SmoothWeightKernel<Scalar> >(t), _refPos(refPos), _dz(dz) { }

	MULTIARCH inline Scalar w(const VectorType& q, const MyPoint&  attributes) const
	{
		Scalar d  = (attributes.spos()-_refPos).norm();
		const Scalar dz = attributes.dz();
		if (d > _t || dz > _dz)
			return Scalar(0.);
		return _wk.f(d/_t);
	}
private:
	ScreenVectorType _refPos;
	float _dz;
};
//! [w_def]

//! [fit_def]
typedef Grenaille::Basket<MyPoint,ProjectWeightFunc,Grenaille::OrientedSphereFit, Grenaille::GLSParam> Gls;
//! [fit_def]

//! [data_acces]
__device__ int getId(const int x,
					 const int y,
					 const int width,
					 const int height,
					 const int component,
					 const int nbComponent)
{
	return (component) + nbComponent*(x + y * width);
}

__device__ VectorType getVector(const int x,
								const int y,
								const int width,
								const int height,
								const float * buffer)
{
	VectorType r;
	r << Scalar(buffer[getId(x,y,width,height,0,3)]),
		Scalar(buffer[getId(x,y,width,height,1,3)]),
		Scalar(buffer[getId(x,y,width,height,2,3)]);
	return r;
}


//! [data_acces]


//! [kernel]
__global__ void doGLS_kernel(  int* params, //[w, h, scale, nbQueries]
							 float* queries,
							 float *positions,
							 float *normals,
							 float* result)
{

	unsigned int ptid = blockIdx.x*blockDim.x + threadIdx.x;

	if (ptid < params[3])
	{
		const int &width     = params[0];
		const int &height    = params[1];
		const int &scale     = params[2];

		// cast float coordinates
		int x = queries[2*ptid];
		int y = queries[2*ptid + 1];

		ScreenVectorType refPos;
		refPos << x, y;


		int dx, dy; // neighbor offset ids
		int nx, ny; // neighbor ids

		Gls gls;
		gls.setWeightFunc(ProjectWeightFunc(scale, refPos));
		gls.init( getVector(x,y,width,height,positions) );

		if ( getVector(x,y,width,height,normals).squaredNorm() == 0.f ){
			result[getId(x,y,width,height,0,1)] = -1.0;
		}
		else{
			//result[getId(x,y,width,height,0,1)] = getVector(x,y,width,height,normals)(0);
			VectorType p, n;

			// collect neighborhood
			VectorType one = VectorType::Zero();

			for(dy = -scale; dy != scale; dy++)
				for(dx = -scale; dx != scale; dx++){
					nx = x+dx;
					ny = y+dy;


					// Check image boundaries
					if (nx >= 0 && ny >= 0 && nx < width && ny < height){
						n = getVector(nx,ny,width,height,normals);

						// add nei only when the normal is properly defined
						// need to use an explicit floating point comparison with pycuda
						if ( n.squaredNorm() != 0.f ) {

							// RGB to XYZ remapping
							n =  2.f * n - one;
							n.normalize();

							// GLS computation
							gls.addNeighbor(MyPoint(getVector(nx,ny,width,height,positions),
								n,
								ScreenVectorType(nx,ny)));
						}
					}
				}
				// closed form minimization
				gls.finalize();
				result[getId(x,y,width,height,0,1)] = gls.kappa();
		}
	}
}
//! [kernel]

/**
* \brief RGB basic color representation
*/
typedef struct {
	double r,g,b;
}Color;

/**
* \brief Return Color corresponding to the value param. Simulating a "seismic" like color map
*/
__host__ Color getColor(double value, double valueMin, double valueMax)
{
	Color c = {1.0, 1.0, 1.0};
	double dv;

	// Unknown values in our kernel
	if(value == -1.)
	{
		return c;
	}

	// Threshold
	if (value < valueMin)
	{
		value = valueMin;
	}

	if (value > valueMax)
	{
		value = valueMax;
	}

	// Interval
	dv = valueMax - valueMin;

	// Seismic color map like
	if(value < (valueMin + 0.5 * dv))
	{
		c.r = 2 * (value - valueMin) / dv;
		c.g = 2 * (value - valueMin) / dv;
		c.b = 1;
	}
	else
	{
		c.b = 2 - 2 * (value - valueMin) / dv;
		c.g = 2 - 2 * (value - valueMin) / dv;
		c.r = 1;
	}

	return c;
}

/**
* \brief Load input images with freeimageplus lib
*/
__host__ bool loadImages(fipImage& positions, fipImage& normals, const char* positionsFilename, const char* normalsFilename)
{
	if(!positions.load(positionsFilename))
	{
		fprintf(stderr, "Cannot load positions\n");
		return 0;
	}

	if(!normals.load(normalsFilename))
	{
		fprintf(stderr, "Cannot load normal map\n");
		return 0;
	}

	positions.convertTo24Bits();
	normals.convertTo24Bits();

	return 1;
}

/**
* \brief Init input datas to be used on host
*/
__host__ bool initInputDatas(const fipImage& positions, const fipImage& normals, float** positionsInfos, float** normalsInfos,
							 unsigned int& width, unsigned int& height)
{
	BYTE* positionsPixels = 0;
	positionsPixels = positions.accessPixels();
	if(!positionsPixels)
	{
		fprintf(stderr, "Cannot get positions datas\n");
		return 0;
	}

	BYTE* normalsPixels = 0;
	normalsPixels = normals.accessPixels();
	if(!normalsPixels)
	{
		fprintf(stderr, "Cannot get normals datas\n");
		return 0;
	}

	width = positions.getWidth();
	height = positions.getHeight();

	(*positionsInfos) = new float[width*height*3];
	(*normalsInfos) = new float[width*height*3];
	if(!*positionsInfos || !*normalsInfos)
	{
		fprintf(stderr, "Cannot alloc memory in initInputDatas\n");
		return 0;
	}

	for(int i = 0; i < width * height; ++i)
	{
		(*positionsInfos)[i * 3 + 0] = positionsPixels[i * 3 + 0] / 255.f * 2.f - 1.f;
		(*positionsInfos)[i * 3 + 1] = positionsPixels[i * 3 + 1] / 255.f * 2.f - 1.f;
		(*positionsInfos)[i * 3 + 2] = positionsPixels[i * 3 + 2] / 255.f * 2.f - 1.f;

		(*normalsInfos)[i * 3 + 0] = normalsPixels[i * 3 + 0] / 255.f;
		(*normalsInfos)[i * 3 + 1] = normalsPixels[i * 3 + 1] / 255.f;
		(*normalsInfos)[i * 3 + 2] = normalsPixels[i * 3 + 2] / 255.f;
	}

	positionsPixels = 0;
	normalsPixels = 0;

	return 1;
}

bool initQueries(const unsigned int& width, const unsigned int& height, float** queries, int& nbQueries)
{
	nbQueries = width * height;
	(*queries) = new float[width*height*2];

	if(!*queries)
	{
		fprintf(stderr, "Cannot alloc memory in initQueries\n");
		return 0;
	}

	for(int y = 0; y < height; ++y)
	{
		for(int x = 0; x < width; ++x)
		{
			(*queries)[2 * (x + y * width)] = x;
			(*queries)[2 * (x + y * width) + 1] = y;
		}
	}

	return 1;
}

/**
* \brief Save results into png image
*/
__host__ bool saveResult(float* results, const unsigned int& width, const unsigned int& height,
						 const char* positionsFilename, const char* resultFilename)
{
	float kappaMin = *std::min_element(results, results + width*height);
	float kappaMax = *std::max_element(results, results + width*height);
	std::cout << "Kappa min : " << kappaMin << std::endl;
	std::cout << "Kappa max : " << kappaMax << std::endl;

	fipImage result;
	if(!result.load(positionsFilename))
	{
		fprintf(stderr, "Cannot load positions\n");
		return 0;
	}

	result.convertTo24Bits();

	BYTE* resultInfos = 0;
	resultInfos = result.accessPixels();
	if(!resultInfos)
	{
		fprintf(stderr, "Cannot get result datas\n");
		return 0;
	}

	for(int i = 0; i < width * height; ++i)
	{
		//check nan
		if(results[i] != results[i])
		{
			results[i] = 0.f;
		}

		Color c = getColor(results[i], -10., 10.);

		resultInfos[i * 3 + 0] = c.r * 255.;
		resultInfos[i * 3 + 1] = c.g * 255.;
		resultInfos[i * 3 + 2] = c.b * 255.;
		//resultInfos[i * 4 + 3] = 255.;
	}

	if(!result.save(resultFilename, 0))
	{
		fprintf(stderr, "Cannot save image\n");
	}

	resultInfos = 0;
	result.clear();

	return 1;
}

int main()
{
	std::string positionsFilename = "./data/ssgls_sample_wc.png";
	std::string normalsFilename = "./data/ssgls_sample_normal.png";
	std::string resultFilename = "./data/ssgls_sample_results.png";

	fipImage positions, normals;

	if(!loadImages(positions, normals, positionsFilename.c_str(), normalsFilename.c_str()))
	{
		return 0;
	}

	float fScale = 10.f;
	unsigned int width = 0;
	unsigned int height = 0;
	float* positionsInfos = 0;
	float* normalsInfos = 0;

	if(!initInputDatas(positions, normals, &positionsInfos, &normalsInfos, width, height))
	{
		return 0;
	}

	std::cout << "Image size : " << width << "*" << height << std::endl;

	float *queries = 0;
	int nbQueries;
	if(!initQueries(width, height, &queries, nbQueries))
	{
		return 0;
	}

	std::cout << "Nb queries : " << nbQueries << std::endl;

	/*********** Init Output ************/
	float *results = new float[width*height];
	for(int i = 0; i < width * height; ++i)
	{
		results[i] = 0.f;
	}

	/************* Init device mem *************/
	size_t sizeResults = width * height * sizeof(float);
	size_t sizeImg = width * height * 3 * sizeof(float);
	int *params = new int[4];
	params[0] = width;
	params[1] = height;
	params[2] = (int)fScale;
	params[3] = nbQueries;

	float* positionsInfos_device;
	float* normalsInfos_device;
	float* results_device;
	float* queries_device;
	int* params_device;

	CUDA_CHECK_RETURN( hipMalloc(&positionsInfos_device, sizeImg) );
	CUDA_CHECK_RETURN( hipMemcpy(positionsInfos_device, positionsInfos, sizeImg, hipMemcpyHostToDevice) );

	CUDA_CHECK_RETURN( hipMalloc(&normalsInfos_device, sizeImg) );
	CUDA_CHECK_RETURN( hipMemcpy(normalsInfos_device, normalsInfos, sizeImg, hipMemcpyHostToDevice) );

	CUDA_CHECK_RETURN( hipMalloc(&queries_device, sizeResults*2) );
	CUDA_CHECK_RETURN( hipMemcpy(queries_device, queries, sizeResults*2, hipMemcpyHostToDevice) );

	CUDA_CHECK_RETURN( hipMalloc(&params_device, 4 * sizeof(int)) );
	CUDA_CHECK_RETURN( hipMemcpy(params_device, params, 4 * sizeof(int), hipMemcpyHostToDevice) );

	CUDA_CHECK_RETURN( hipMalloc(&results_device, sizeResults) );
	CUDA_CHECK_RETURN( hipMemcpy(results_device, results, sizeResults, hipMemcpyHostToDevice) );

	/************* Memory conf *************/

	int numThreadsPerBlock = 128;
	int numBlocks = nbQueries / numThreadsPerBlock;
	if((nbQueries % numThreadsPerBlock) > 0)
	{
		numBlocks += 1;
	}

	dim3 dimGrid(numBlocks, 1);
	dim3 dimBlock(numThreadsPerBlock, 1, 1);

	/************* Kernel Call *************/

	std::cout << "ssCurvature running..." << std::endl;

	doGLS_kernel<<<dimGrid, dimBlock>>>(params_device, queries_device, positionsInfos_device, normalsInfos_device, results_device);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());

	std::cout << "ssCurvature completed..." << std::endl;

	/************* Get Results *************/
	CUDA_CHECK_RETURN( hipMemcpy(results, results_device, sizeResults, hipMemcpyDeviceToHost) );

	std::cout << "Finalizing..." << std::endl;

	/********** Cuda Free ************/
	CUDA_CHECK_RETURN( hipFree(positionsInfos_device) );
	CUDA_CHECK_RETURN( hipFree(normalsInfos_device) );
	CUDA_CHECK_RETURN( hipFree(results_device) );
	CUDA_CHECK_RETURN( hipFree(queries_device) );
	CUDA_CHECK_RETURN( hipFree(params_device) );

	/********** Saving result ************/
	if(!saveResult(results, width, height, positionsFilename.c_str(), resultFilename.c_str()))
	{
		return 0;
	}

	/********** Free Memory *********/
	positions.clear();
	normals.clear();

	delete [] positionsInfos;
	delete [] normalsInfos;
	delete [] queries;
	delete [] results;
	delete [] params;

	CUDA_CHECK_RETURN(hipDeviceReset());

	std::cout << "Finished !" << std::endl;

	return 0;
}
